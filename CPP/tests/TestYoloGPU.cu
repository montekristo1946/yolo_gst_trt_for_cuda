#include <vector>
#include <iostream>
#include <vector>
#include <iostream>
#include <fstream>
#include <iterator>
#include "YoloToolsGPU.h"
#include <chrono>


using namespace std;


std::vector<float> ReadFileToVector(const std::string &filename) {
    std::vector<float> result;
    std::ifstream inputFile(filename);
    if (inputFile.is_open()) {
        std::copy(std::istream_iterator<float>(inputFile),
                  std::istream_iterator<float>(),
                  std::back_inserter(result));
        inputFile.close();
    } else {
        std::cerr << "Unable to open file " << filename << std::endl;
    }
    return result;
}


void TestNmsGPU() {
    hipStream_t _stream;
    auto resCudaStreamCreate = hipStreamCreate(&_stream);
    if (resCudaStreamCreate != 0) {
        runtime_error("[TRTEngine::InitTRTEngine]  Not CudaStreamCreate {}");
    }

    vector<float> resultDl = ReadFileToVector("../examples/img_001_result.txt");

    float *ml_ptr_device = nullptr;
    hipMallocAsync((void **) &ml_ptr_device, resultDl.size() * sizeof(float), _stream);
    auto resultCudaMemcopyHostToDevice = hipMemcpy(ml_ptr_device,
                                                    resultDl.data(),
                                                    resultDl.size() * sizeof(float),
                                                    hipMemcpyHostToDevice);
    if (resultCudaMemcopyHostToDevice != 0) {
        throw runtime_error(" [hipMalloc] resCudaMalloc false");
    }

    int shapeBboxinOutLayer = 8400;
    float kConfThresh = 0.1;
    float kNmsThresh = 0.6f;
    const int countLabel = 4;
    int kMaxNumOutputBbox = 1000;

    auto _yoloToolsGPU = new YoloToolsGPU(shapeBboxinOutLayer,
                                          kConfThresh,
                                          kMaxNumOutputBbox,
                                          countLabel,
                                          kNmsThresh,
                                          _stream);

    for (int i = 0; i < 10000000; ++i) {
        auto start = chrono::system_clock::now();

        auto rects = _yoloToolsGPU->GetDetctionsBbox((float *) ml_ptr_device);

        auto checkCudaStreamSynchronize = hipStreamSynchronize(_stream);

        if (checkCudaStreamSynchronize != 0) {
            throw runtime_error("[hipMalloc] checkCudaStreamSynchronize false");
        }

        if (rects.size() != 17) {
            cout<<rects.size()<<endl;
            throw runtime_error("[GetDetctionsBbox] fail detect");

        }
        auto endAllProcess = chrono::system_clock::now();

        cout << "iter: " << i
             << " Nms time: " << chrono::duration_cast<chrono::microseconds>(endAllProcess - start).count()<< " microseconds"
             << endl;
    }
    hipFree(ml_ptr_device);
    _yoloToolsGPU->~YoloToolsGPU();
}


int main(int argc, char *argv[]) {
    TestNmsGPU();
    return 0;
}
